#include "hip/hip_runtime.h"

#include <bitset>
#define _USE_MATH_DEFINES
#include <cmath>
#include <hip/hip_runtime.h>
#include <cuda/hip/hip_vector_types.h>
#include <hipcub/hipcub.hpp>
#include <numeric>
#include <string>
#include <sstream>
#include <type_traits>
#include <vector>
#include "hdi/utils/log_helper_functions.h"
#include "hdi/dimensionality_reduction/gpgpu_sne/bvh/utils/assert.h"
#include "hdi/dimensionality_reduction/gpgpu_sne/bvh/bvh.h"
#include "hdi/dimensionality_reduction/gpgpu_sne/bvh/bvh_kern.h"
#include <cuda_gl_interop.h> // include last because of OpenGL header

template <unsigned D>
using vec = std::conditional<D == 2, float2, float4>::type;

template <typename genType> 
inline
genType ceilDiv(genType n, genType div) {
  return (n + div - 1) / div;
}

namespace hdi {
  namespace dr {  
    namespace bvh {
      template <unsigned D>
      BVH<D>::BVH()
      : _isInit(false)
      { }

      template <unsigned D>
      BVH<D>::~BVH() {
        if (_isInit) {
          destr();
        }
      }

      template <unsigned D>
      void BVH<D>::init(const TsneParameters &params, GLuint posBuffer, GLuint boundsBuffer, unsigned nPos) {
        _params = params;
        _layout = BVHLayout(2, nPos);

        // Fire up hipcub::RadixSort to figure out temporary memory size in bytes
        size_t tempSize;
        hipcub::DeviceRadixSort::SortPairs<uint, float4>(
          nullptr, tempSize, nullptr, nullptr, nullptr, nullptr, _layout.nPos
        );
        
        // Initialize managing memory subclasses
        _intMemr.init(_layout, tempSize);
        _extMemr.init(_layout);
        _extPos.init(posBuffer, InteropType::eReadOnly);
        _extBounds.init(boundsBuffer, InteropType::eReadOnly);

        // Fill unsorted indices
        {
          std::vector<uint> idx(nPos);
          std::iota(idx.begin(), idx.end(), 0u);
          hipMemcpy(_intMemr.ptr(BVHIntMemr::MemrType::eIdxIn),
            idx.data(), sizeof(uint) * idx.size(), hipMemcpyHostToDevice);
        }
        
        // Set root node range, as this (REALLY) shouldn't change
        _extMemr.map();
        {
          float4 rootNode = make_float4(0.f, 0.f, 0.f, static_cast<float>(_layout.nPos));
          float4 rootDiam = make_float4(0.f, 0.f, 0.f, 0.f);
          hipMemcpy(_extMemr.ptr(BVHExtMemr<D>::MemrType::eNode),
            &rootNode, sizeof(float4), hipMemcpyHostToDevice);
          hipMemcpy(_extMemr.ptr(BVHExtMemr<D>::MemrType::eDiam),
            &rootDiam, sizeof(float4), hipMemcpyHostToDevice);
        }
        _extMemr.unmap();

        for (auto &timer : _timers) {
          timer.init();
        }
        _isInit = true;
      }

      template <unsigned D>
      void BVH<D>::destr() {
        _extPos.destr();
        _extBounds.destr();
        _extMemr.destr();
        _intMemr.destr();
        for (auto &timer : _timers) {
          timer.destr();
        }
        _isInit = false;
      }

      template <unsigned D>
      void BVH<D>::compute(bool rebuild, unsigned iteration) {
        using vec = vec<D>;
        
        // Map external resources for access
        _extMemr.map();
        _extPos.map();
        _extBounds.map();

        // bool rebuild = iteration <= _params._remove_exaggeration_iter || iteration % 8 == 0;
        
        // Generate 30-bit morton codes over unsorted embedding positions
        _timers[TIMR_MORTON].tick();
        if (rebuild) {
          kernConstrMorton<D><<<ceilDiv(_layout.nPos, 256u), 256u>>>(
            _layout,
            (BVHBounds<D> *) _extBounds.ptr(),
            (vec *) _extPos.ptr(),
            (uint *) _intMemr.ptr(BVHIntMemr::MemrType::eMortonIn)
          );
        }
        _timers[TIMR_MORTON].tock();

        // Perform radix sort on embedding positions using 30-bit morton codes
        _timers[TIMR_SORT].tick();
        if (rebuild) {
          size_t tempSize = _intMemr.memrSize(BVHIntMemr::MemrType::eTemp);
          const int msb = 30;
          const int lsb = msb - _layout.nLvls;
          hipcub::DeviceRadixSort::SortPairs<uint, uint>(
            (void *) _intMemr.ptr(BVHIntMemr::MemrType::eTemp), 
            tempSize, 
            (uint *) _intMemr.ptr(BVHIntMemr::MemrType::eMortonIn),
            (uint *) _intMemr.ptr(BVHIntMemr::MemrType::eMortonOut),
            (uint *) _intMemr.ptr(BVHIntMemr::MemrType::eIdxIn),
            (uint *) _intMemr.ptr(BVHIntMemr::MemrType::eIdxOut),
            (int) _layout.nPos, 
            lsb, msb
          );
          // Construct sorted position list from unsorted positions and sorted indices
          kernConstrPos<D><<<ceilDiv(_layout.nPos, 256u), 256u>>>(
            _layout,
            (uint *) _intMemr.ptr(BVHIntMemr::MemrType::eIdxOut),
            (vec *) _extPos.ptr(), 
            (vec *) _extMemr.ptr(BVHExtMemr<D>::MemrType::ePos)
          );
        }
        _timers[TIMR_SORT].tock();

        _timers[TIMR_SUBDIV].tick();
        if (rebuild) {
          // Perform subdivision and construct leaf nodes
          for (uint l = 1; l < _layout.nLvls; l++) {
            const uint begin = 1 << l;
            const uint end = 2 * begin;
            kernConstrSubdiv<D><<<ceilDiv(end - begin, 256u), 256u>>>(
              _layout,
              l, _layout.nLvls, begin, end,
              (uint *) _intMemr.ptr(BVHIntMemr::MemrType::eMortonOut),
              (vec *) _extMemr.ptr(BVHExtMemr<D>::MemrType::ePos),
              (float4 *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eNode),
              (vec *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eMinB),
              (float4 *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eDiam)
            );
          }
        }
        _timers[TIMR_SUBDIV].tock();
        
        // Compute additional tree data bottom-up
        _timers[TIMR_DATA].tick();
        // {
        //   const uint begin = 1 << (_layout.nLvls - 1);
        //   const uint end = 2 * begin;
        //   kernConstrLeaf<D><<<ceilDiv(_layout.nNodes, 256u), 256u>>>(
        //     _layout,
        //     begin, end,
        //     (uint *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eIdx),
        //     (vec *) _extMemr.ptr(BVHExtMemr<D>::MemrType::ePos),
        //     (float4 *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eNode),
        //     (vec *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eMinB),
        //     (float4 *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eDiam)
        //   );
        // }

        for (int l = _layout.nLvls - 1; l > 0; l -= 6) {
          const uint begin = 1 << l;
          const uint end = 2 * begin;
          kernConstrBbox<D><<<ceilDiv(end - begin, 64u), 64u>>>(
            _layout, 
            l, _layout.nLvls, begin, end,
            (vec *) _extMemr.ptr(BVHExtMemr<D>::MemrType::ePos),
            (float4 *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eNode),
            (vec *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eMinB),
            (float4 *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eDiam)
          );
        }
        _timers[TIMR_DATA].tock();

        _timers[TIMR_CLEANUP].tick();
        /* kernConstrCleanup<D><<<ceilDiv(_layout.nNodes, 1024u), 1024u>>>(
          _layout.nNodes,
          (float4 *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eNode),
          (vec *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eMinB),
          (float4 *) _extMemr.ptr(BVHExtMemr<D>::MemrType::eDiam)
        ); */
        _timers[TIMR_CLEANUP].tock();

        if (iteration == 0) {
          std::vector<float4> node0Buffer(1);
          std::vector<float4> node1Buffer(1);

          hipMemcpy(node0Buffer.data(), _extMemr.ptr(BVHExtMemr<D>::MemrType::eNode),
            1 * sizeof(float4), hipMemcpyDeviceToHost);
          hipMemcpy(node1Buffer.data(), _extMemr.ptr(BVHExtMemr<D>::MemrType::eDiam),
            1 * sizeof(float4), hipMemcpyDeviceToHost);

          for (int i = 0; i < 16; i++) {
            // std::cout << std::bitset<32>(codeBuffer[i]) << '\n';
            std::cout << "Range\t"
                      << node1Buffer[i].w << '-' << node0Buffer[i].w << '\n';
            // std::cout << "Center\t"
            //           << node0Buffer[i].x << ", "
            //           << node0Buffer[i].y << ", "
            //           << node0Buffer[i].z << '\n';
          }
        }

        // Unmap external resources for access
        _timers[TIMR_TEST].tick();
        _extMemr.unmap();
        _extBounds.unmap();
        _extPos.unmap();
        _timers[TIMR_TEST].tock();
        
        for (auto &timer : _timers) {
          timer.poll();
        }

        if (iteration >= static_cast<unsigned>(_params._iterations) - 1) {
          utils::secureLog(_logger, "\nBVH construction");
          CU_LOG_TIMER(_logger, _timers[TIMR_MORTON], "  Morton");
          CU_LOG_TIMER(_logger, _timers[TIMR_SORT], "  Sorting");
          CU_LOG_TIMER(_logger, _timers[TIMR_SUBDIV], "  Subdiv");
          CU_LOG_TIMER(_logger, _timers[TIMR_DATA], "  Bboxes");
          CU_LOG_TIMER(_logger, _timers[TIMR_CLEANUP], "  Cleanup");
          CU_LOG_TIMER(_logger, _timers[TIMR_TEST ], "  Test");
        }
      }
      
      // Explicit template instantiations
      template class BVH<2>;
      template class BVH<3>;
    }
  }
}